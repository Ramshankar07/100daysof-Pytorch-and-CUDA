#include "hip/hip_runtime.h"
//C coalesced memory access is important for parallel sum

#include <iostream>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    extern __shared__ float sharedMemory[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;
    if (i < N) {

        sharedMemory[tid] = A[i] + B[i+blockDim.x];
        _syncthreads();
        for (int stride = 1; stride < blockDim.x; stride *= 2) {
            int temp = 0;
            if (tid >= stride) {
                temp = sharedMemory[tid - stride];
            }
            __syncthreads();
            sharedMemory[tid] += temp;
            __syncthreads();
        }

        // Write result to global memory
        C[index] = sharedMemory[tid];
    
    }
}

int main() {
    const int N = 10;
    float A[N], B[N], C[N];

    float *d_a, *d_b,*d_c;
    hipMalloc(&d_a,N*sizeof(float));//allocate memory on device
    hipMalloc(&d_b,N*sizeof(float));
    hipMalloc(&d_c,N*sizeof(float));
    hipMemcpy(d_a,A,N*sizeof(float),hipMemcpyHostToDevice);//copy data from host to device
    hipMemcpy(d_b,B,N*sizeof(float),hipMemcpyHostToDevice);
    int blocksize=256;//number of threads in a block
    int gridsize=ceil(N/blocksize);//number of blocks in a grid
    vectorAdd<<<gridsize,blocksize>>>(d_a,d_b,d_c,N);//call the kernel function to perform vector addition
    hipMemcpy(C,d_c,N*sizeof(float),hipMemcpyDeviceToHost);//copy data from device to host
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}
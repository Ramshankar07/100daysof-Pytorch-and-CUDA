#include "hip/hip_runtime.h"
//C coalesced memory access is important for parallel sum

#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

__global__ void LayerNorm(const float* A, const float* B, int N) {
    
    int tid = threadIdx.x;
    float *sum = &row[blockDim.x];
    int i = blockIdx.x * blockDim.x + tid;
    if (i < N) {
        extern __shared__ float sharedMem[];
        float *row = sharedMem;
        for (int col = threadIdx.y; col < N; col += blockDim.y) {
            row_data[col] = A[row * N + col];
        }
        _syncthreads();
        float mean = 0.0f;
        for (int col = 0; col < N; col++) {
            mean += row_data[col];
        }
        mean /= N;

        // Compute variance
        float variance = 0.0f;
        for (int col = 0; col < N; col++) {
            variance += (row_data[col] - mean) * (row_data[col] - mean);
        }
        variance /= N;
        float stddev = sqrtf(variance + 1e-7);

        // Normalize
        for (int col = threadIdx.y; col < N; col += blockDim.y) {
            B[row * N + col] = (row_data[col] - mean) / stddev;
        }
    }
}
int main() {
    const int N = 10;

    float *A = new float[N * N];
    float *B = new float[N * N];
    for (int i = 0; i < N * N; i++) {
         A[i] = static_cast<float>(rand()) / RAND_MAX;//random values between 0 and 1

    }
    float *d_a, *d_b;
    hipMalloc(&d_a,N*N*sizeof(float));//allocate memory on device
    hipMalloc(&d_b,N*N*sizeof(float));
    hipMemcpy(d_a,A,N*N*sizeof(float),hipMemcpyHostToDevice);
    int blocksize=256;//number of threads in a block
    size_t sharedMemory = (N+2*blocksize)*sizeof(float)//number of blocks in a grid
    LayerNorm<<<N,blocksize,sharedMemory>>>(d_a,d_b,N);//call the kernel function to perform vector addition
    hipMemcpy(B,d_b,N*sizeof(float),hipMemcpyDeviceToHost);//copy data from device to host
    hipFree(d_a);
    hipFree(d_b);

}
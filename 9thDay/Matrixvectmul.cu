#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__device__ float randomFunction(float x, float y)
{
    return x * y;
}

__global__ void matrixFunction(const float *A, const float *B, float *C, const int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size)
    {
        C[i + size * j] = randomFunction(A[i + size * j], B[ j]);
    }
}

int main()
{   
    int N = 8;
    int BLOCK_SIZE = 64;
    dim3 blockDim(BLOCK_SIZE * BLOCK_SIZE);
    dim3 gridDim(N + BLOCK_SIZE - 1 / BLOCK_SIZE, N + BLOCK_SIZE - 1 / BLOCK_SIZE);
    int size = sizeof(float) * N * N;

    float *A,*B,*C;
    float *dA,*dB,*dC;
    A = new float[N*N];
    B = new float[N]; //since we are doing matrix vector multiplication
    C = new float[N];
    size_t sizeB = sizeof(float) * N;
    hipMalloc((void**)&dA,size);
    hipMalloc((void**)&dB,sizeB);
    hipMalloc((void**)&dC,size);

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            A[i + N * j] = 1.0f; 
            
        }
        B[i] = 2.0f;
        C[i] = 0.0f;
    }
    
    hipMemcpy(dA,A,size,hipMemcpyHostToDevice);
    hipMemcpy(dB,B,size,hipMemcpyHostToDevice);

    // now we have everything set up
    matrixFunction<<<gridDim,blockDim>>>(dA,dB,dC,N);
    hipDeviceSynchronize();

    hipMemcpy(C,dC,size,hipMemcpyDeviceToHost);

    for (int i = 0; i < N*N; i++) {
        std::cout << C[i] << " ";
        if ((i + 1) % N == 0) std::cout << std::endl;
    }
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}
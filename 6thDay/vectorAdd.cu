
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int N = 10;
    float A[N], B[N], C[N];

    float *d_a, *d_b,*d_c;
    hipMalloc(&d_a,N*sizeof(float));//allocate memory on device
    hipMalloc(&d_b,N*sizeof(float));
    hipMalloc(&d_c,N*sizeof(float));
    hipMemcpy(d_a,A,N*sizeof(float),hipMemcpyHostToDevice);//copy data from host to device
    hipMemcpy(d_b,B,N*sizeof(float),hipMemcpyHostToDevice);
    int blocksize=256;//number of threads in a block
    int gridsize=ceil(N/blocksize);//number of blocks in a grid
    vectorAdd<<<gridsize,blocksize>>>(d_a,d_b,d_c,N);//call the kernel function to perform vector addition
    hipMemcpy(C,d_c,N*sizeof(float),hipMemcpyDeviceToHost);//copy data from device to host
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}
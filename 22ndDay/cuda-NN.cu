#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipDNN.h>
#include <crand.h>
// This code won't have backpropagation and grads, infernence : That's tomorrow's work
#define CheckCudaError(err) { 
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }

}

#define CheckCUDNNError(err) { 
    hipdnnStatus_t err = (func);
    if(err != HIPDNN_STATUS_SUCCESS) {
        std::cerr<<"CUDNN Error: " << hipdnnGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

const int batch_size = 8;
const int input_size = 600;
const int hidden_size = 1000;
const int output_size = 10;

const float learning_rate = 0.01;
const int num_epochs = 4;


void init_weights(float *weights, int size) {
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 12345);
    hiprandGenerateUniform(gen, weights, size);
    hiprandDestroyGenerator(gen);
}
void init_bias(float *bias, int size) {
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 12345);
    hiprandGenerateUniform(gen, bias, size);
    hiprandDestroyGenerator(gen);
}
int main(){

    hipdnnHandle_t handle;
    CheckCUDNNError(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t input_desc, hidden_desc, hidden_desc2, output_desc;
    CheckCUDNNError(hipdnnCreateTensorDescriptor(&input_desc));
    CheckCUDNNError(hipdnnCreateTensorDescriptor(&hidden_desc));
    CheckCUDNNError(hipdnnCreateTensorDescriptor(&hidden_desc2));
    CheckCUDNNError(hipdnnCreateTensorDescriptor(&output_desc));

    CheckCUDNNError(hipdnnSetTensor4dDescriptor(input_desc,HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, input_size, 1, 1));
    CheckCUDNNError(hipdnnSetTensor4dDescriptor(hidden_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, hidden_size, 1, 1));
    CheckCUDNNError(hipdnnSetTensor4dDescriptor(hidden_desc2, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, hidden_size, 1, 1));
    CheckCUDNNError(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, output_size, 1, 1));
    
    
    hipdnnActivationDescriptor_t relu_desc;
    CheckCUDNNError(hipdnnCreateActivationDescriptor(&relu_desc));
    CheckCUDNNError(hipdnnSetActivationDescriptor(relu_desc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0f));
    
    hipdnnFilterDescriptor_t weight_desc, weight_desc2, weight_desc3;
    CheckCUDNNError(hipdnnCreateFilterDescriptor(&weight_desc));
    CheckCUDNNError(hipdnnCreateFilterDescriptor(&weight_desc2));
    CheckCUDNNError(hipdnnCreateFilterDescriptor(&weight_desc3));
    CheckCUDNNError(hipdnnSetFilter4dDescriptor(weight_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, hidden_size, input_size, 1, 1));
    CheckCUDNNError(hipdnnSetFilter4dDescriptor(weight_desc2, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, hidden_size, hidden_size, 1, 1));
    CheckCUDNNError(hipdnnSetFilter4dDescriptor(weight_desc3, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, output_size, hidden_size, 1, 1));
    
    hipdnnConvolutionDescriptor_t conv_desc, conv_desc2;
    CheckCUDNNError(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CheckCUDNNError(hipdnnCreateConvolutionDescriptor(&conv_desc2));
    CheckCUDNNError(hipdnnSetConvolution2dDescriptor(conv_desc, 1, 1, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
    CheckCUDNNError(hipdnnSetConvolution2dDescriptor(conv_desc2, 1, 1, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    //mem alloc for NN weights and bias
    float *weights, *weights2, *weights3;
    float *bias, *bias2, *bias3;
    size_t weights_size = hidden_size * input_size * sizeof(float);
    size_t weights2_size = hidden_size * hidden_size * sizeof(float);
    size_t weights3_size = output_size * hidden_size * sizeof(float);
    size_t bias_size = hidden_size * sizeof(float);
    size_t bias2_size = hidden_size * sizeof(float);
    size_t bias3_size = output_size * sizeof(float);
    CheckCudaError(hipMalloc(&weights, weights_size));
    CheckCudaError(hipMalloc(&weights2, weights2_size));
    CheckCudaError(hipMalloc(&weights3, weights3_size));
    CheckCudaError(hipMalloc(&bias, bias_size));
    CheckCudaError(hipMalloc(&bias2, bias2_size));
    CheckCudaError(hipMalloc(&bias3, bias3_size));
    CheckCudaError(hipMemset(weights, 0, weights_size));
    CheckCudaError(hipMemset(weights2, 0, weights2_size));
    CheckCudaError(hipMemset(weights3, 0, weights3_size));
    CheckCudaError(hipMemset(bias, 0, bias_size));
    CheckCudaError(hipMemset(bias2, 0, bias2_size));
    CheckCudaError(hipMemset(bias3, 0, bias3_size));
    //mem alloc for NN input, hidden, hidden2, output
    float *input, *hidden, *hidden2, *output;
    size_t input_size = batch_size * input_size * sizeof(float);
    size_t hidden_size = batch_size * hidden_size * sizeof(float);
    size_t hidden2_size = batch_size * hidden_size * sizeof(float);
    size_t output_size = batch_size * output_size * sizeof(float);
    
    CheckCudaError(hipMalloc(&input, batch_size*input_size*sizeof(float)));
    CheckCudaError(hipMalloc(&hidden, batch_size*hidden_size*sizeof(float)));
    CheckCudaError(hipMalloc(&hidden2, batch_size*hidden2_size*sizeof(float)));
    CheckCudaError(hipMalloc(&output, batch_size*output_size*sizeof(float)));
    
    //with dummy data
    init_weights(weights, weights_size);
    init_weights(weights2, weights2_size);
    init_weights(weights3, weights3_size);
    init_bias(bias, bias_size);
    init_bias(bias2, bias2_size);
    init_bias(bias3, bias3_size);
    init_weights(input, input_size);
    init_weights(output, output_size);

    //training loop
    for(int i=0; i<num_epochs;i++){

        float alpha= 1.0f; float beta= 0.0f;
        //first layer
        CheckCUDNNError(hipdnnConvolutionForward(handle, &alpha, input_desc, input, weight_desc, weights, conv_desc, conv_desc, &beta, hidden_desc, hidden));
        CheckCUDNNError(hipdnnActivationForward(handle, relu_desc, &alpha, hidden_desc, hidden, &beta, hidden_desc2, hidden2));
        //second layer
        CheckCUDNNError(hipdnnConvolutionForward(handle, &alpha, hidden_desc2, hidden2, weight_desc2, weights2, conv_desc2, conv_desc2, &beta, hidden_desc, hidden));
        CheckCUDNNError(hipdnnActivationForward(handle, relu_desc, &alpha, hidden_desc, hidden, &beta, hidden_desc2, hidden2));
        //third layer
        CheckCUDNNError(hipdnnConvolutionForward(handle, &alpha, hidden_desc2, hidden2, weight_desc3, weights3, conv_desc2, conv_desc2, &beta, output_desc, output));
        CheckCUDNNError(hipdnnActivationForward(handle, relu_desc, &alpha, hidden_desc, hidden, &beta, hidden_desc2, hidden2));

       
        Loss=0.0f;
        for(int j=0; j<batch_size; j++){
            Loss+=fabs(output[j*output_size] - output[j*output_size]);

            }
        CheckCudaError(hipMemcpy(input, output, batch_size*output_size*sizeof(float), hipMemcpyDeviceToHost));
        std::cout << "Epoch: " << i << " Loss: " << Loss << std::endl;
            
    }
    CheckCudaError(hipFree(weights));
    CheckCudaError(hipFree(weights2));
    CheckCudaError(hipFree(weights3));
    CheckCudaError(hipFree(bias));
    CheckCudaError(hipFree(bias2));
    CheckCudaError(hipFree(bias3));
    CheckCudaError(hipFree(input));
    CheckCudaError(hipFree(hidden));
    CheckCudaError(hipFree(hidden2));
    CheckCudaError(hipFree(output));
    CheckCUDNNError(hipdnnDestroy(handle));
    CheckCUDNNError(hipdnnDestroyTensorDescriptor(input_desc));
    CheckCUDNNError(hipdnnDestroyTensorDescriptor(hidden_desc));
    CheckCUDNNError(hipdnnDestroyTensorDescriptor(hidden_desc2));
    CheckCUDNNError(hipdnnDestroyTensorDescriptor(output_desc));
    CheckCUDNNError(hipdnnDestroyActivationDescriptor(relu_desc));
    CheckCUDNNError(hipdnnDestroyFilterDescriptor(weight_desc));
    CheckCUDNNError(hipdnnDestroyFilterDescriptor(weight_desc2));
    CheckCUDNNError(hipdnnDestroyFilterDescriptor(weight_desc3));
    CheckCUDNNError(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CheckCUDNNError(hipdnnDestroyConvolutionDescriptor(conv_desc2));
    return 0;




}   


